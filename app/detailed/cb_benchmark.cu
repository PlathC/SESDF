#include "hip/hip_runtime.h"
#include <array>
#include <memory>

#ifdef _WIN32
#include <Windows.h>
#endif // _WIN32

#include <GL/gl3w.h>

//
#include <ContourBuildup/BufferObject.hpp>
#include <ContourBuildup/Texture2D.hpp>
#include <ContourBuildup/particleSystem.cuh>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include <vector_functions.h>

#include "bcs/core/molecule.hpp"
#include "cb_benchmark.cuh"

// This function returns the best GPU (with maximum GFLOPS)
int cudaUtilGetMaxGflopsDeviceId()
{
    int device_count = 0;
    hipGetDeviceCount( &device_count );

    hipDeviceProp_t device_properties;
    int            max_gflops_device = 0;
    int            max_gflops        = 0;

    int current_device = 0;
    hipGetDeviceProperties( &device_properties, current_device );
    max_gflops = device_properties.multiProcessorCount * device_properties.clockRate;
    ++current_device;

    while ( current_device < device_count )
    {
        hipGetDeviceProperties( &device_properties, current_device );
        int gflops = device_properties.multiProcessorCount * device_properties.clockRate;
        if ( gflops > max_gflops )
        {
            max_gflops        = gflops;
            max_gflops_device = current_device;
        }
        ++current_device;
    }

    return max_gflops_device;
}

namespace bcs
{
    CBBenchmark cbBenchmark( ConstSpan<Vec4f> molecule )
    {
        constexpr float        probeRadius        = 1.4f;
        constexpr uint32_t     gridDimensions     = 16;
        constexpr unsigned int atomNeighborCount  = 64;
        const bcs::Aabb        aabb               = bcs::getAabb( molecule );
        const unsigned int     probeNeighborCount = 32;

        static bool setCUDAGLDevice = false;
        if ( setCUDAGLDevice )
        {
            cudaGLSetGLDevice( cudaUtilGetMaxGflopsDeviceId() );
#ifndef NDEBUG
            printf( "cudaGLSetGLDevice: %s\n", hipGetErrorString( hipGetLastError() ) );
#endif // NDEBUG
            setCUDAGLDevice = false;
        }

        uint numAtoms = molecule.size;

        // set grid dimensions
        glm::uvec3 gridSize     = glm::uvec3( gridDimensions );
        uint       numGridCells = gridSize.x * gridSize.y * gridSize.z;

        glm::vec3 size         = aabb.max - aabb.min;
        glm::vec3 worldSize    = glm::vec3( size.x, size.y, size.z );
        uint      gridSortBits = 18; // increase this for larger grids

        // set parameters
        SimParams params;
        params.gridSize  = make_uint3( gridSize.x, gridSize.y, gridSize.z );
        params.numCells  = numGridCells;
        params.numBodies = numAtoms;
        // this->params.worldOrigin = make_float3(-1.0f, -1.0f, -1.0f);

        params.worldOrigin = make_float3( aabb.min.x, aabb.min.y, aabb.min.z );
        params.cellSize = make_float3( worldSize.x / gridSize.x, worldSize.y / gridSize.y, worldSize.z / gridSize.z );
        params.probeRadius     = probeRadius;
        params.maxNumNeighbors = atomNeighborCount;

        // allocate host storage
        // hPos_.clear();
        // hPos_.resize( this->numAtoms, glm::vec4( 0.0f ) );

        GLuint sphereVAO_            = 0;
        GLuint torusVAO_             = 0;
        GLuint sphericalTriangleVAO_ = 0;

        // GPU data
        float * m_dPos;
        float * m_dSortedPos;
        float * m_dSortedProbePos;
        uint *  m_dNeighborCount;
        uint *  m_dNeighbors;
        float * m_dSmallCircles;
        uint *  m_dSmallCircleVisible;
        uint *  m_dSmallCircleVisibleScan;
        float * m_dArcs;
        uint *  m_dArcIdxK;
        uint *  m_dArcCount;
        uint *  m_dArcCountScan;

        // grid data for sorting method
        uint * m_dGridParticleHash;  // grid hash value for each particle
        uint * m_dGridParticleIndex; // particle index for each particle
        uint * m_dGridProbeHash;     // grid hash value for each probe
        uint * m_dGridProbeIndex;    // particle index for each probe
        uint * m_dCellStart;         // index of start of each cell in sorted list
        uint * m_dCellEnd;           // index of end of cell

        enum class Buffers : GLuint
        {
            PROBE_POS         = 0,
            SPHERE_TRIA_VEC_1 = 1,
            SPHERE_TRIA_VEC_2 = 2,
            SPHERE_TRIA_VEC_3 = 3,
            TORUS_POS         = 4,
            TORUS_VS          = 5,
            TORUS_AXIS        = 6,
            SING_TEX          = 7,
            TEX_COORD         = 8,
            ATOM_POS          = 9,
            BUFF_COUNT        = 10
        };
        std::array<std::unique_ptr<glowl::BufferObject>, static_cast<int>( Buffers::BUFF_COUNT )> buffers_;

        // singularity texture
        std::unique_ptr<glowl::Texture2D> singTex_;

        // maximum number of probe neighbors
        unsigned int texHeight;
        unsigned int texWidth;
        unsigned int width;
        unsigned int height;

        // allocate GPU data
        unsigned int memSize = sizeof( float ) * 4 * numAtoms;
        // array for atom positions
        allocateArray( (void **)&m_dPos, memSize );
        // hipMalloc(  (void**)&m_dPos, memSize);
        // hipError_t e;
        // e = hipGetLastError();

        // hipMemGetInfo
        // uint free, total;
        // hipMemGetInfo( &free, &total);
        // megamol::core::utility::log::Log::DefaultLog.WriteMsg( megamol::core::utility::log::Log::LEVEL_ERROR,
        //     "Free GPU Memory: %i / %i (MB)", free / ( 1024 * 1024), total / ( 1024 * 1024));
        //  array for sorted atom positions
        allocateArray( (void **)&m_dSortedPos, memSize );
        // array for sorted atom positions
        allocateArray( (void **)&m_dSortedProbePos, memSize * atomNeighborCount );
        // array for the counted number of atoms
        allocateArray( (void **)&m_dNeighborCount, numAtoms * sizeof( uint ) );
        // array for the neighbor atoms
        allocateArray( (void **)&m_dNeighbors, numAtoms * atomNeighborCount * sizeof( uint ) );
        // array for the small circles
        allocateArray( (void **)&m_dSmallCircles, numAtoms * atomNeighborCount * 4 * sizeof( float ) );
        // array for the small circle visibility
        allocateArray( (void **)&m_dSmallCircleVisible, numAtoms * atomNeighborCount * sizeof( uint ) );
        // array for the small circle visibility prefix sum
        allocateArray( (void **)&m_dSmallCircleVisibleScan, numAtoms * atomNeighborCount * sizeof( uint ) );

        // Avoid overflow during calculation
        const std::size_t arcCount = numAtoms * atomNeighborCount * atomNeighborCount;
        // array for the arcs
        allocateArray( (void **)&m_dArcs, arcCount * 4 * sizeof( float ) );
        // array for the arcs
        allocateArray( (void **)&m_dArcIdxK, arcCount * sizeof( uint ) );
        // array for the arc count
        allocateArray( (void **)&m_dArcCount, numAtoms * atomNeighborCount * sizeof( uint ) );
        // array for the arc count scan (prefix sum)
        allocateArray( (void **)&m_dArcCountScan, numAtoms * atomNeighborCount * sizeof( uint ) );

        allocateArray( (void **)&m_dGridParticleHash, numAtoms * sizeof( uint ) );
        allocateArray( (void **)&m_dGridParticleIndex, numAtoms * sizeof( uint ) );

        allocateArray( (void **)&m_dGridProbeHash, numAtoms * atomNeighborCount * sizeof( uint ) );
        allocateArray( (void **)&m_dGridProbeIndex, numAtoms * atomNeighborCount * sizeof( uint ) );

        allocateArray( (void **)&m_dCellStart, numGridCells * sizeof( uint ) );
        allocateArray( (void **)&m_dCellEnd, numGridCells * sizeof( uint ) );

        // clear all buffers
        for ( auto & e : buffers_ )
        {
            e = nullptr;
        }

        // re-create all buffers
        buffers_[ static_cast<int>( Buffers::PROBE_POS ) ] = std::make_unique<glowl::BufferObject>(
            GL_ARRAY_BUFFER, nullptr, numAtoms * atomNeighborCount * 4 * sizeof( float ), GL_DYNAMIC_DRAW );
        cudaGLRegisterBufferObject( buffers_[ static_cast<int>( Buffers::PROBE_POS ) ]->getName() );
        getLastCudaError( "init failed" );

        buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_1 ) ] = std::make_unique<glowl::BufferObject>(
            GL_ARRAY_BUFFER, nullptr, numAtoms * atomNeighborCount * 4 * sizeof( float ), GL_DYNAMIC_DRAW );
        cudaGLRegisterBufferObject( buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_1 ) ]->getName() );

        buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_2 ) ] = std::make_unique<glowl::BufferObject>(
            GL_ARRAY_BUFFER, nullptr, numAtoms * atomNeighborCount * 4 * sizeof( float ), GL_DYNAMIC_DRAW );
        cudaGLRegisterBufferObject( buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_2 ) ]->getName() );

        buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_3 ) ] = std::make_unique<glowl::BufferObject>(
            GL_ARRAY_BUFFER, nullptr, numAtoms * atomNeighborCount * 4 * sizeof( float ), GL_DYNAMIC_DRAW );
        cudaGLRegisterBufferObject( buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_3 ) ]->getName() );

        buffers_[ static_cast<int>( Buffers::TORUS_POS ) ] = std::make_unique<glowl::BufferObject>(
            GL_ARRAY_BUFFER, nullptr, numAtoms * atomNeighborCount * 4 * sizeof( float ), GL_DYNAMIC_DRAW );
        cudaGLRegisterBufferObject( buffers_[ static_cast<int>( Buffers::TORUS_POS ) ]->getName() );

        buffers_[ static_cast<int>( Buffers::TORUS_VS ) ] = std::make_unique<glowl::BufferObject>(
            GL_ARRAY_BUFFER, nullptr, numAtoms * atomNeighborCount * 4 * sizeof( float ), GL_DYNAMIC_DRAW );
        cudaGLRegisterBufferObject( buffers_[ static_cast<int>( Buffers::TORUS_VS ) ]->getName() );

        buffers_[ static_cast<int>( Buffers::TORUS_AXIS ) ] = std::make_unique<glowl::BufferObject>(
            GL_ARRAY_BUFFER, nullptr, numAtoms * atomNeighborCount * 4 * sizeof( float ), GL_DYNAMIC_DRAW );
        cudaGLRegisterBufferObject( buffers_[ static_cast<int>( Buffers::TORUS_AXIS ) ]->getName() );

        // get maximum texture size
        GLint texSize;
        glGetIntegerv( GL_MAX_TEXTURE_SIZE, &texSize );
        texHeight      = std::min( numAtoms * 3, static_cast<uint>( texSize ) );
        texWidth       = probeNeighborCount * ( ( numAtoms * 3 ) / texSize + 1 );
        params.texSize = texSize;

        // create singularity texture
        std::vector<std::pair<GLenum, GLint>>   int_parameters = { { GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE },
                                                                   { GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE },
                                                                   { GL_TEXTURE_MIN_FILTER, GL_NEAREST },
                                                                   { GL_TEXTURE_MAG_FILTER, GL_NEAREST } };
        std::vector<std::pair<GLenum, GLfloat>> float_parameters;
        glowl::TextureLayout                    tx_layout { GL_RGB32F,
                                         static_cast<int>( texWidth ),
                                         static_cast<int>( texHeight ),
                                         1,
                                         GL_RGB,
                                         GL_FLOAT,
                                         1,
                                         int_parameters,
                                         float_parameters };
        singTex_ = std::make_unique<glowl::Texture2D>( "molecule_cbc_singTex", tx_layout, nullptr );

        // create PBO
        buffers_[ static_cast<int>( Buffers::SING_TEX ) ] = std::make_unique<glowl::BufferObject>(
            GL_PIXEL_UNPACK_BUFFER, nullptr, texWidth * texHeight * 3 * sizeof( float ), GL_DYNAMIC_DRAW );
        cudaGLRegisterBufferObject( buffers_[ static_cast<int>( Buffers::SING_TEX ) ]->getName() );

        // create texture coordinate buffer object
        buffers_[ static_cast<int>( Buffers::TEX_COORD ) ] = std::make_unique<glowl::BufferObject>(
            GL_ARRAY_BUFFER, nullptr, numAtoms * 3 * 3 * sizeof( float ), GL_DYNAMIC_DRAW );
        cudaGLRegisterBufferObject( buffers_[ static_cast<int>( Buffers::TEX_COORD ) ]->getName() );

        // set parameters
        setParameters( &params );

        // create VAOs
        if ( sphericalTriangleVAO_ == 0 )
        {
            glGenVertexArrays( 1, &sphericalTriangleVAO_ );
        }
        glBindVertexArray( sphericalTriangleVAO_ );

        buffers_[ static_cast<int>( Buffers::PROBE_POS ) ]->bind();
        glEnableVertexAttribArray( 0 );
        glVertexAttribPointer( 0, 4, GL_FLOAT, GL_FALSE, 0, nullptr );

        buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_1 ) ]->bind();
        glEnableVertexAttribArray( 1 );
        glVertexAttribPointer( 1, 4, GL_FLOAT, GL_FALSE, 0, nullptr );

        buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_2 ) ]->bind();
        glEnableVertexAttribArray( 2 );
        glVertexAttribPointer( 2, 4, GL_FLOAT, GL_FALSE, 0, nullptr );

        buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_3 ) ]->bind();
        glEnableVertexAttribArray( 3 );
        glVertexAttribPointer( 3, 4, GL_FLOAT, GL_FALSE, 0, nullptr );

        buffers_[ static_cast<int>( Buffers::TEX_COORD ) ]->bind();
        glEnableVertexAttribArray( 4 );
        glVertexAttribPointer( 4, 3, GL_FLOAT, GL_FALSE, 0, nullptr );

        glBindVertexArray( 0 );
        glBindBuffer( GL_ARRAY_BUFFER, 0 );
        glDisableVertexAttribArray( 0 );
        glDisableVertexAttribArray( 1 );
        glDisableVertexAttribArray( 2 );
        glDisableVertexAttribArray( 3 );
        glDisableVertexAttribArray( 4 );

        if ( torusVAO_ == 0 )
        {
            glGenVertexArrays( 1, &torusVAO_ );
        }
        glBindVertexArray( torusVAO_ );

        buffers_[ static_cast<int>( Buffers::TORUS_POS ) ]->bind();
        glEnableVertexAttribArray( 0 );
        glVertexAttribPointer( 0, 4, GL_FLOAT, GL_FALSE, 0, nullptr );

        buffers_[ static_cast<int>( Buffers::TORUS_AXIS ) ]->bind();
        glEnableVertexAttribArray( 1 );
        glVertexAttribPointer( 1, 4, GL_FLOAT, GL_FALSE, 0, nullptr );

        buffers_[ static_cast<int>( Buffers::TORUS_VS ) ]->bind();
        glEnableVertexAttribArray( 2 );
        glVertexAttribPointer( 2, 4, GL_FLOAT, GL_FALSE, 0, nullptr );

        glBindVertexArray( 0 );
        glBindBuffer( GL_ARRAY_BUFFER, 0 );
        glDisableVertexAttribArray( 0 );
        glDisableVertexAttribArray( 1 );
        glDisableVertexAttribArray( 2 );

        // Benchmark timing
        CBBenchmark timings {};

        // CPU -> GPU
        if ( buffers_[ static_cast<int>( Buffers::ATOM_POS ) ] == nullptr )
        {
            buffers_[ static_cast<int>( Buffers::ATOM_POS ) ] = std::make_unique<glowl::BufferObject>(
                GL_ARRAY_BUFFER, molecule.ptr, molecule.size * 4 * sizeof( float ), GL_DYNAMIC_DRAW );
            cudaGLRegisterBufferObject( buffers_[ static_cast<int>( Buffers::ATOM_POS ) ]->getName() );
        }
        else
        {
            buffers_[ static_cast<int>( Buffers::ATOM_POS ) ]->rebuffer( molecule.ptr,
                                                                         molecule.size * 4 * sizeof( float ) );
        }

        if ( sphereVAO_ == 0 )
        {
            glGenVertexArrays( 1, &sphereVAO_ );
            glBindVertexArray( sphereVAO_ );

            buffers_[ static_cast<int>( Buffers::ATOM_POS ) ]->bind();
            glEnableVertexAttribArray( 0 );
            glVertexAttribPointer( 0, 4, GL_FLOAT, GL_FALSE, 0, nullptr );

            glBindVertexArray( 0 );
            glBindBuffer( GL_ARRAY_BUFFER, 0 );
            glDisableVertexAttribArray( 0 );
        }

        // update constants
        params.probeRadius = probeRadius;
        setParameters( &params );

        // map OpenGL buffer object for writing from CUDA
        float * atomPosPtr;
        cudaGLMapBufferObject( (void **)&atomPosPtr, buffers_[ static_cast<int>( Buffers::ATOM_POS ) ]->getName() );

        {
            TimingEvent accelerationStructureTimer {};
            accelerationStructureTimer.start();

            // calculate grid hash
            calcHash( m_dGridParticleHash,
                      m_dGridParticleIndex,
                      // m_dPos,
                      atomPosPtr,
                      numAtoms );

            // sort particles based on hash
            sortParticles( m_dGridParticleHash, m_dGridParticleIndex, numAtoms );

            // reorder particle arrays into sorted order and
            // find start and end of each cell
            reorderDataAndFindCellStart( m_dCellStart,
                                         m_dCellEnd,
                                         m_dSortedPos,
                                         m_dGridParticleHash,
                                         m_dGridParticleIndex,
                                         // m_dPos,
                                         atomPosPtr,
                                         numAtoms,
                                         numGridCells );

            accelerationStructureTimer.stop();
            timings.circle += accelerationStructureTimer.getElapsedMs();
        }

        // unmap buffer object
        cudaGLUnmapBufferObject( buffers_[ static_cast<int>( Buffers::ATOM_POS ) ]->getName() );

        {
            TimingEvent neighborTimer {};
            neighborTimer.start();

            // find neighbors of all atoms and compute small circles
            findNeighborsCB( m_dNeighborCount,
                             m_dNeighbors,
                             m_dSmallCircles,
                             m_dSortedPos,
                             m_dCellStart,
                             m_dCellEnd,
                             numAtoms,
                             atomNeighborCount,
                             numGridCells );

            // find and remove unnecessary small circles
            removeCoveredSmallCirclesCB( m_dSmallCircles,
                                         m_dSmallCircleVisible,
                                         m_dNeighborCount,
                                         m_dNeighbors,
                                         m_dSortedPos,
                                         numAtoms,
                                         params.maxNumNeighbors );

            neighborTimer.stop();
            timings.circle += neighborTimer.getElapsedMs();
        }

        hipMemset( m_dArcCount, 0, numAtoms * atomNeighborCount * sizeof( uint ) );
        {
            TimingEvent intersectionTimer {};
            intersectionTimer.start();

            // compute all arcs for all small circles
            computeArcsCB( m_dSmallCircles,
                           m_dSmallCircleVisible,
                           m_dNeighborCount,
                           m_dNeighbors,
                           m_dSortedPos,
                           m_dArcs,
                           m_dArcCount,
                           numAtoms,
                           params.maxNumNeighbors );

            // ---------- vertex buffer object generation (for rendering) ----------
            // count total number of small circles
            scanParticles( m_dSmallCircleVisible, m_dSmallCircleVisibleScan, numAtoms * atomNeighborCount );

            intersectionTimer.stop();
            timings.intersection += intersectionTimer.getElapsedMs();
        }

        // get total number of small circles
        uint numSC  = 0;
        uint lastSC = 0;
        checkCudaErrors( hipMemcpy( (void *)&numSC,
                                     (void *)( m_dSmallCircleVisibleScan + ( numAtoms * atomNeighborCount ) - 1 ),
                                     sizeof( uint ),
                                     hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy( (void *)&lastSC,
                                     (void *)( m_dSmallCircleVisible + ( numAtoms * atomNeighborCount ) - 1 ),
                                     sizeof( uint ),
                                     hipMemcpyDeviceToHost ) );
        numSC += lastSC;

        {
            TimingEvent scanTimer {};
            scanTimer.start();

            // count total number of arcs
            scanParticles( m_dArcCount, m_dArcCountScan, numAtoms * atomNeighborCount );

            scanTimer.stop();
            timings.intersection += scanTimer.getElapsedMs();
        }

        // get total number of probes
        uint numProbes    = 0;
        uint lastProbeCnt = 0;
        checkCudaErrors( hipMemcpy( (void *)&numProbes,
                                     (void *)( m_dArcCountScan + ( numAtoms * atomNeighborCount ) - 1 ),
                                     sizeof( uint ),
                                     hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy( (void *)&lastProbeCnt,
                                     (void *)( m_dArcCount + ( numAtoms * atomNeighborCount ) - 1 ),
                                     sizeof( uint ),
                                     hipMemcpyDeviceToHost ) );
        numProbes += lastProbeCnt;

        // resize torus buffer objects
        cudaGLUnregisterBufferObject( buffers_[ static_cast<int>( Buffers::TORUS_POS ) ]->getName() );
        cudaGLUnregisterBufferObject( buffers_[ static_cast<int>( Buffers::TORUS_VS ) ]->getName() );
        cudaGLUnregisterBufferObject( buffers_[ static_cast<int>( Buffers::TORUS_AXIS ) ]->getName() );
        buffers_[ static_cast<int>( Buffers::TORUS_POS ) ]->rebuffer( nullptr, numSC * 4 * sizeof( float ) );
        buffers_[ static_cast<int>( Buffers::TORUS_VS ) ]->rebuffer( nullptr, numSC * 4 * sizeof( float ) );
        buffers_[ static_cast<int>( Buffers::TORUS_AXIS ) ]->rebuffer( nullptr, numSC * 4 * sizeof( float ) );
        cudaGLRegisterBufferObject( buffers_[ static_cast<int>( Buffers::TORUS_POS ) ]->getName() );
        cudaGLRegisterBufferObject( buffers_[ static_cast<int>( Buffers::TORUS_VS ) ]->getName() );
        cudaGLRegisterBufferObject( buffers_[ static_cast<int>( Buffers::TORUS_AXIS ) ]->getName() );

        // resize probe buffer object
        cudaGLUnregisterBufferObject( buffers_[ static_cast<int>( Buffers::PROBE_POS ) ]->getName() );
        buffers_[ static_cast<int>( Buffers::PROBE_POS ) ]->rebuffer( nullptr, numProbes * 4 * sizeof( float ) );
        cudaGLRegisterBufferObject( buffers_[ static_cast<int>( Buffers::PROBE_POS ) ]->getName() );

        // resize spherical triangle buffer objects
        cudaGLUnregisterBufferObject( buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_1 ) ]->getName() );
        cudaGLUnregisterBufferObject( buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_2 ) ]->getName() );
        cudaGLUnregisterBufferObject( buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_3 ) ]->getName() );
        buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_1 ) ]->rebuffer( nullptr,
                                                                              numProbes * 4 * sizeof( float ) );
        buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_2 ) ]->rebuffer( nullptr,
                                                                              numProbes * 4 * sizeof( float ) );
        buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_3 ) ]->rebuffer( nullptr,
                                                                              numProbes * 4 * sizeof( float ) );
        cudaGLRegisterBufferObject( buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_1 ) ]->getName() );
        cudaGLRegisterBufferObject( buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_2 ) ]->getName() );
        cudaGLRegisterBufferObject( buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_3 ) ]->getName() );

        // map probe buffer object for writing from CUDA
        float * probePosPtr;
        cudaGLMapBufferObject( (void **)&probePosPtr, buffers_[ static_cast<int>( Buffers::PROBE_POS ) ]->getName() );

        // map spherical triangle buffer objects for writing from CUDA
        float *sphereTriaVec1Ptr, *sphereTriaVec2Ptr, *sphereTriaVec3Ptr;
        cudaGLMapBufferObject( (void **)&sphereTriaVec1Ptr,
                               buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_1 ) ]->getName() );
        cudaGLMapBufferObject( (void **)&sphereTriaVec2Ptr,
                               buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_2 ) ]->getName() );
        cudaGLMapBufferObject( (void **)&sphereTriaVec3Ptr,
                               buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_3 ) ]->getName() );

        // map torus buffer objects for writing from CUDA
        float *torusPosPtr, *torusVSPtr, *torusAxisPtr;
        cudaGLMapBufferObject( (void **)&torusPosPtr, buffers_[ static_cast<int>( Buffers::TORUS_POS ) ]->getName() );
        cudaGLMapBufferObject( (void **)&torusVSPtr, buffers_[ static_cast<int>( Buffers::TORUS_VS ) ]->getName() );
        cudaGLMapBufferObject( (void **)&torusAxisPtr, buffers_[ static_cast<int>( Buffers::TORUS_AXIS ) ]->getName() );

        {
            TimingEvent writingTimer {};
            writingTimer.start();

            // compute vertex buffer objects for probe positions
            writeProbePositionsCB( probePosPtr,
                                   sphereTriaVec1Ptr,
                                   sphereTriaVec2Ptr,
                                   sphereTriaVec3Ptr,
                                   torusPosPtr,
                                   torusVSPtr,
                                   torusAxisPtr,
                                   m_dNeighborCount,
                                   m_dNeighbors,
                                   m_dSortedPos,
                                   m_dArcs,
                                   m_dArcCount,
                                   m_dArcCountScan,
                                   m_dSmallCircleVisible,
                                   m_dSmallCircleVisibleScan,
                                   m_dSmallCircles,
                                   numAtoms,
                                   atomNeighborCount );

            writingTimer.stop();
            timings.intersection += writingTimer.getElapsedMs();
        }

        // unmap torus buffer objects
        cudaGLUnmapBufferObject( buffers_[ static_cast<int>( Buffers::TORUS_POS ) ]->getName() );
        cudaGLUnmapBufferObject( buffers_[ static_cast<int>( Buffers::TORUS_VS ) ]->getName() );
        cudaGLUnmapBufferObject( buffers_[ static_cast<int>( Buffers::TORUS_AXIS ) ]->getName() );

        // unmap spherical triangle buffer objects
        cudaGLUnmapBufferObject( buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_1 ) ]->getName() );
        cudaGLUnmapBufferObject( buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_2 ) ]->getName() );
        cudaGLUnmapBufferObject( buffers_[ static_cast<int>( Buffers::SPHERE_TRIA_VEC_3 ) ]->getName() );

        {
            TimingEvent singularityTimer {};
            singularityTimer.start();

            // ---------- singularity handling ----------
            // calculate grid hash
            calcHash( m_dGridProbeHash, m_dGridProbeIndex, probePosPtr, numProbes );

            // sort probes based on hash
            sortParticles( m_dGridProbeHash, m_dGridProbeIndex, numProbes );

            // reorder particle arrays into sorted order and find start and end of each cell
            reorderDataAndFindCellStart( m_dCellStart,
                                         m_dCellEnd,
                                         m_dSortedProbePos,
                                         m_dGridProbeHash,
                                         m_dGridProbeIndex,
                                         probePosPtr,
                                         numProbes,
                                         numGridCells );

            singularityTimer.stop();
            timings.neighborhood += singularityTimer.getElapsedMs();
        }

        // unmap probe buffer object
        cudaGLUnmapBufferObject( buffers_[ static_cast<int>( Buffers::PROBE_POS ) ]->getName() );

        // resize texture coordinate buffer object
        cudaGLUnregisterBufferObject( buffers_[ static_cast<int>( Buffers::TEX_COORD ) ]->getName() );
        cudaGLUnregisterBufferObject( buffers_[ static_cast<int>( Buffers::SING_TEX ) ]->getName() );
        buffers_[ static_cast<int>( Buffers::TEX_COORD ) ]->rebuffer( nullptr, numProbes * 3 * sizeof( float ) );
        buffers_[ static_cast<int>( Buffers::SING_TEX ) ]->rebuffer(
            nullptr, numProbes * probeNeighborCount * 3 * sizeof( float ) );
        cudaGLRegisterBufferObject( buffers_[ static_cast<int>( Buffers::TEX_COORD ) ]->getName() );
        cudaGLRegisterBufferObject( buffers_[ static_cast<int>( Buffers::SING_TEX ) ]->getName() );

        // map texture coordinate buffer object for writing from CUDA
        float * texCoordPtr;
        cudaGLMapBufferObject( (void **)&texCoordPtr, buffers_[ static_cast<int>( Buffers::TEX_COORD ) ]->getName() );
        // map singularity texture buffer object for writing from CUDA
        float * singTexPtr;
        cudaGLMapBufferObject( (void **)&singTexPtr, buffers_[ static_cast<int>( Buffers::SING_TEX ) ]->getName() );

        {
            TimingEvent singularityTimer {};
            singularityTimer.start();

            // find all intersecting probes for each probe and write them to a texture
            writeSingularityTextureCB( texCoordPtr,
                                       singTexPtr,
                                       m_dSortedProbePos,
                                       m_dGridProbeIndex,
                                       m_dCellStart,
                                       m_dCellEnd,
                                       numProbes,
                                       probeNeighborCount,
                                       numGridCells );

            singularityTimer.stop();
            timings.neighborhood += singularityTimer.getElapsedMs();
        }

        // copyArrayFromDevice( m_hPos, m_dSortedProbePos, 0, sizeof(float)*4);
        // std::cout << "probe: " << m_hPos[0] << ", " << m_hPos[1] << ", " << m_hPos[2] << " r = " << m_hPos[3]
        // << std::endl; copyArrayFromDevice( m_hPos, singTexPtr, 0, sizeof(float)*3*this->probeNeighborCount);
        // for( unsigned int i = 0; i < this->probeNeighborCount; i++ ) {
        //     std::cout << "neighbor probe " << i << ": " << m_hPos[i*3] << " " << m_hPos[i*3+1] << " " <<
        //     m_hPos[i*3+2] << std::endl;
        // }

        // unmap texture coordinate buffer object
        cudaGLUnmapBufferObject( buffers_[ static_cast<int>( Buffers::TEX_COORD ) ]->getName() );
        // unmap singularity texture buffer object
        cudaGLUnmapBufferObject( buffers_[ static_cast<int>( Buffers::SING_TEX ) ]->getName() );

        // copy PBO to texture
        buffers_[ static_cast<int>( Buffers::SING_TEX ) ]->bind();
        // glEnable( GL_TEXTURE_2D );
        singTex_->bindTexture();
        glTexSubImage2D( GL_TEXTURE_2D,
                         0,
                         0,
                         0,
                         ( numProbes / params.texSize + 1 ) * probeNeighborCount,
                         numProbes % params.texSize,
                         GL_RGB,
                         GL_FLOAT,
                         NULL );
        glBindTexture( GL_TEXTURE_2D, 0 );
        // glDisable( GL_TEXTURE_2D );
        glBindBuffer( GL_PIXEL_UNPACK_BUFFER, 0 );

        freeArray( m_dPos );
        freeArray( m_dSortedPos );
        freeArray( m_dSortedProbePos );
        freeArray( m_dNeighborCount );
        freeArray( m_dNeighbors );
        freeArray( m_dSmallCircles );
        freeArray( m_dSmallCircleVisible );
        freeArray( m_dSmallCircleVisibleScan );
        freeArray( m_dArcs );
        freeArray( m_dArcIdxK );
        freeArray( m_dArcCount );
        freeArray( m_dArcCountScan );
        freeArray( m_dGridParticleHash );
        freeArray( m_dGridParticleIndex );
        freeArray( m_dGridProbeHash );
        freeArray( m_dGridProbeIndex );
        freeArray( m_dCellStart );
        freeArray( m_dCellEnd );

        // Added to avoid "Already mapped" error
        for ( const auto & e : buffers_ )
        {
            cudaGLUnregisterBufferObject( e->getName() );
        }

        return timings;
    }
} // namespace bcs
